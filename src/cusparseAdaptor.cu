#include "cusparseAdaptor.hu"
#include <iostream>

using namespace thundercat;

CusparseAdaptor* thundercat::newCusparseAdaptor() {
  return new CusparseAdaptor();
};

void thundercat::deleteCusparseAdaptor(CusparseAdaptor* adaptor) {
  delete adaptor;
}

CusparseAdaptor::~CusparseAdaptor() {
  hipFree(rowIndexDevPtr);
  hipFree(colIndexDevPtr);
  hipFree(valDevPtr);

  hipFree(x);
  hipFree(y);

  hipsparseDestroyMatDescr(descr);
  hipsparseDestroy(handle);
}

void CusparseAdaptor::init() {
  handle = 0;
  descr = 0;

  hipsparseCreate(&handle);

  hipsparseStatus_t status = hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
}

void CusparseAdaptor::preprocess(int nnz, int m, int n, int * rowPtr, int* colIdx, double* values) {

  M = m;
  N = n;
  NNZ = nnz;

  hipError_t error = hipMalloc((void**)&rowIndexDevPtr, (N + 1) * sizeof(int));
  error = hipMalloc((void**)&colIndexDevPtr, NNZ * sizeof(int));
  error = hipMalloc((void**)&valDevPtr, NNZ * sizeof(double));

  error = hipMemcpy((void *)rowIndexDevPtr, (void*)rowPtr, (size_t) ((N + 1) * sizeof(int)), hipMemcpyHostToDevice);
  error = hipMemcpy((void*)colIndexDevPtr, (void*)colIdx, (size_t) (NNZ * sizeof(int)), hipMemcpyHostToDevice);
  error = hipMemcpy((void*) valDevPtr, (void*)values, (size_t) (NNZ * sizeof(double)), hipMemcpyHostToDevice);

  error = hipMalloc((void**)&x, M * sizeof(double));
  error = hipMalloc((void**)&y, N * sizeof(double));
  error = hipMemset(y, 0, N * sizeof(double));
}

void CusparseAdaptor::setX(double *v) {
  hipMemcpy((void*) x, (void*) v,(size_t)(M*sizeof(double)),hipMemcpyHostToDevice);
  hipDeviceSynchronize();
}

void CusparseAdaptor::getY(double *w) {
  hipMemcpy((void*) w, (void*) y,(size_t)(N*sizeof(double)),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

void CusparseAdaptor::spmv() {
  double alpha = 1.0;
  double beta = 1.0;
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M, N, NNZ, &alpha,
                 descr, valDevPtr, rowIndexDevPtr, colIndexDevPtr, x, &beta, y);

  hipDeviceSynchronize();
}