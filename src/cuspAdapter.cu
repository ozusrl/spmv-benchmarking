#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include "cuspAdapter.hu"

using namespace thundercat;
void CuspAdapter::preprocess( int m, int n, int nnz, int * rowPtr, int * colIndx, double * values) {
  M = m;
  N = n;
  NNZ = nnz;

  int *devRowPtr;
  int *devColIndx;
  double *devValues;

  hipMalloc(&devRowPtr, (N+1) * sizeof(int));
  hipMalloc(&devColIndx, NNZ * sizeof(int));
  hipMalloc(&devValues, NNZ * sizeof(double));
  hipMalloc(&devX, M * sizeof(double));
  hipMalloc(&devY, N * sizeof(double));

  hipMemcpy(devRowPtr,  rowPtr, (N+1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devColIndx, colIndx, NNZ * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devValues, values, NNZ * sizeof(double), hipMemcpyHostToDevice);

  // *NOTE* raw pointers must be wrapped with thrust::device_ptr!
  thrust::device_ptr<int>   wrapped_device_Ap(devRowPtr);
  thrust::device_ptr<int>   wrapped_device_Aj(devColIndx);
  thrust::device_ptr<double> wrapped_device_Ax(devValues);
  thrust::device_ptr<double> wrapped_device_x(devX);
  thrust::device_ptr<double> wrapped_device_y(devY);



  DeviceIndexArrayView row_offsets(wrapped_device_Ap, wrapped_device_Ap + N + 1);
  DeviceIndexArrayView column_indices(wrapped_device_Aj, wrapped_device_Aj + NNZ);
  DeviceValueArrayView values_array        (wrapped_device_Ax, wrapped_device_Ax + NNZ);
  DeviceValueArrayView x_local(wrapped_device_x, wrapped_device_x + M);
  DeviceValueArrayView y_local(wrapped_device_y, wrapped_device_y + N);

  DeviceView A_local(M, N, NNZ, row_offsets, column_indices, values_array);
  A = A_local;
  x = x_local;
  y = y_local;

}

void CuspAdapter::setX(double * v) {
  hipMemcpy(devX, v, M * sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
}

void CuspAdapter::getY(double * w) {
  hipMemcpy(w, devY, N * sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

void CuspAdapter::spmv() {
  cusp::multiply(A, x, y);
  hipDeviceSynchronize();
}

CuspAdapter* thundercat::newCuspAdapter() {
  return new CuspAdapter();
}

void thundercat::deleteCuspAdapter(CuspAdapter* handle) {
  delete handle;
}
